#include "hip/hip_runtime.h"
// User: g113@157.88.139.133 
// ExecutionRequest[P:'extinguishing.cu',P:1,T:1,args:'',q:'cudalb'] 
// May 16 2019 13:21:23
#include "cputils.h" // Added by tablon
/*
 * Simplified simulation of fire extinguishing
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2018/2019
 *
 * v1.4
 *
 * (c) 2019 Arturo Gonzalez Escribano
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <cputils.h>

#define RADIUS_TYPE_1		3
#define RADIUS_TYPE_2_3		9
#define THRESHOLD	0.1f

/* Structure to store data of an extinguishing team */
typedef struct {
	int x,y;
	int type;
	int target;
} Team;

/* Structure to store data of a fire focal point */
typedef struct {
	int x,y;
	int start;
	int heat;
	int active; // States: 0 Not yet activated; 1 Active; 2 Deactivated by a team
} FocalPoint;

/* Macro function to simplify accessing with two coordinates to a flattened array */
#define accessMat( arr, exp1, exp2 )	arr[ (exp1) * columns + (exp2) ]

/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s <config_file> | <command_line_args>\n", program_name );
	fprintf(stderr,"\t<config_file> ::= -f <file_name>\n");
	fprintf(stderr,"\t<command_line_args> ::= <rows> <columns> <maxIter> <numTeams> [ <teamX> <teamY> <teamType> ... ] <numFocalPoints> [ <focalX> <focalY> <focalStart> <focalTemperature> ... ]\n");
	fprintf(stderr,"\n");
}

#ifdef DEBUG
/* 
 * Function: Print the current state of the simulation 
 */
void print_status( int iteration, int rows, int columns, float *surface, int num_teams, Team *teams, int num_focal, FocalPoint *focal, float global_residual ) {
	/* 
	 * You don't need to optimize this function, it is only for pretty printing and debugging purposes.
	 * It is not compiled in the production versions of the program.
	 * Thus, it is never used when measuring times in the leaderboard
	 */
	int i,j;

	printf("Iteration: %d\n", iteration );
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	for( i=0; i<rows; i++ ) {
		printf("|");
		for( j=0; j<columns; j++ ) {
			char symbol;
			if ( accessMat( surface, i, j ) >= 1000 ) symbol = '*';
			else if ( accessMat( surface, i, j ) >= 100 ) symbol = '0' + (int)(accessMat( surface, i, j )/100);
			else if ( accessMat( surface, i, j ) >= 50 ) symbol = '+';
			else if ( accessMat( surface, i, j ) >= 25 ) symbol = '.';
			else symbol = '0';

			int t;
			int flag_team = 0;
			for( t=0; t<num_teams; t++ ) 
				if ( teams[t].x == i && teams[t].y == j ) { flag_team = 1; break; }
			if ( flag_team ) printf("[%c]", symbol );
			else {
				int f;
				int flag_focal = 0;
				for( f=0; f<num_focal; f++ ) 
					if ( focal[f].x == i && focal[f].y == j && focal[f].active == 1 ) { flag_focal = 1; break; }
				if ( flag_focal ) printf("(%c)", symbol );
				else printf(" %c ", symbol );
			}
		}
		printf("|\n");
	}
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	printf("Global residual: %f\n\n", global_residual);
}
#endif

/*
 * MAIN PROGRAM
 */
__global__ void init_matrix(float *surface, float *surfaceCopy, int surface_size, int rows, int columns){
	//Compute the thread's gid
	int gid = threadIdx.x + blockDim.x * blockIdx.x;

	//If the id is greater than the surface's size, don't do anything
	if(gid >= surface_size){
		return;
	}

	//Determine to which row and column this thread corresponds to
	int row = gid/columns;
	int column = gid%columns;

	//Initialize the cell
	accessMat(surface, row, column) = 0;
	accessMat(surfaceCopy, row, column) = 0;
}

__global__ void update_heat(float *surface, float *surfaceCopy, int surface_size, int rows, int columns){
	//Compute the thread's gid
	int gid = threadIdx.x + blockDim.x * blockIdx.x;

	//If the id is greater than the surface's size, don't do anything
	if(gid >= surface_size){
		return;
	}

	//Determine to which row and column this thread corresponds to
	int row = gid/columns;
	int column = gid%columns;

	//If this thread is on the border of the surface, don't do anything.
	//Else, add the elements on it's sides and store the result on it's 
	//position to update the heat
	if(row == 0 | row == rows-1 | column == 0 | column == columns-1){
		return;
	}else{
		accessMat(surface, row, column) = ( 
				accessMat( surfaceCopy, row-1, column ) +
				accessMat( surfaceCopy, row+1, column ) +
				accessMat( surfaceCopy, row, column-1 ) +
				accessMat( surfaceCopy, row, column+1 ) ) / 4;
	}			
}

__global__ void reduce_max(float *surface, int reduction_size){
	//Compute the thread's gid
	int gid = threadIdx.x + blockDim.x * blockIdx.x;

	//If the id is greater than half the reduction's size, don't do anything
	if(gid >= reduction_size/2){
		return;
	}

	if(surface[gid] < surface[gid+reduction_size/2]){
		surface[gid] = surface[gid+reduction_size/2];
	}

	//In case the reduction size is odd, there will be a mismatched element. The last thread
	//will have to cover it as well
	if(reduction_size%2 != 0){
		if(gid == 0){
			if(surface[gid] < surface[reduction_size - 1]){
				surface[gid] = surface[reduction_size - 1];
			}
		}
	}
}
__global__ void compute_residual(float *surface, float *surfaceCopy, int surface_size, int rows, int columns){
	//Compute the thread's gid
	int gid = threadIdx.x + blockDim.x * blockIdx.x;

	//Determine to which row and column this thread corresponds to
	int row = gid/columns;
	int column = gid%columns;

	//If this thread is on the border of the surface, initialize the cell
	if(row == 0 | row == rows-1 | column == 0 | column == columns-1){
		surfaceCopy[gid] = 0;
		return;
	}

	//If the id is greater than the surface's size, don't do anything
	if(gid >= surface_size){
		return;
	}

	surfaceCopy[gid] = fabs(surface[gid] - surfaceCopy[gid]);

}

__global__ void update_focal(float *surface, FocalPoint *focal, int num_focal, int columns){
	//Compute the thread's gid
	int gid = threadIdx.x + blockDim.x * blockIdx.x;
	//If the id is greater than the focal point list's size, return
	if(gid >= num_focal){
		return;
	}
	//Else, read the values from the corresponding focal point and update the surface
	if(focal[gid].active == 1){
		//printf("%f\n", accessMat(surface, focal[gid].x, focal[gid].y));
		accessMat(surface, focal[gid].x, focal[gid].y) = focal[gid].heat;
		//printf("%f\n", accessMat(surface, focal[gid].x, focal[gid].y));
	}

}

int main(int argc, char *argv[]) {
	int i,j,t;

	// Simulation data
	int rows, columns, max_iter;
	float *surface, *surfaceCopy;
	int num_teams, num_focal;
	Team *teams;
	FocalPoint *focal;

	/* 1. Read simulation arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc<2) {
		fprintf(stderr,"-- Error in arguments: No arguments\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	int read_from_file = ! strcmp( argv[1], "-f" );
	/* 1.2. Read configuration from file */
	if ( read_from_file ) {
		/* 1.2.1. Open file */
		if (argc<3) {
			fprintf(stderr,"-- Error in arguments: file-name argument missing\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
		FILE *args = cp_abrir_fichero( argv[2] );
		if ( args == NULL ) {
			fprintf(stderr,"-- Error in file: not found: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}	

		/* 1.2.2. Read surface and maximum number of iterations */
		int ok;
		ok = fscanf(args, "%d %d %d", &rows, &columns, &max_iter);
		if ( ok != 3 ) {
			fprintf(stderr,"-- Error in file: reading rows, columns, max_iter from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}

		surface = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );
		surfaceCopy = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );

		if ( surface == NULL || surfaceCopy == NULL ) {
			fprintf(stderr,"-- Error allocating: surface structures\n");
			exit( EXIT_FAILURE );
		}

		/* 1.2.3. Teams information */
		ok = fscanf(args, "%d", &num_teams );
		if ( ok != 1 ) {
			fprintf(stderr,"-- Error file, reading num_teams from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}
		teams = (Team *)malloc( sizeof(Team) * (size_t)num_teams );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_teams; i++ ) {
			ok = fscanf(args, "%d %d %d", &teams[i].x, &teams[i].y, &teams[i].type);
			if ( ok != 3 ) {
				fprintf(stderr,"-- Error in file: reading team %d from file: %s\n", i, argv[1]);
				exit( EXIT_FAILURE );
			}
		}

		/* 1.2.4. Focal points information */
		ok = fscanf(args, "%d", &num_focal );
		if ( ok != 1 ) {
			fprintf(stderr,"-- Error in file: reading num_focal from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}
		focal = (FocalPoint *)malloc( sizeof(FocalPoint) * (size_t)num_focal );
		if ( focal == NULL ) {
			fprintf(stderr,"-- Error allocating: %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_focal; i++ ) {
			ok = fscanf(args, "%d %d %d %d", &focal[i].x, &focal[i].y, &focal[i].start, &focal[i].heat);
			if ( ok != 4 ) {
				fprintf(stderr,"-- Error in file: reading focal point %d from file: %s\n", i, argv[1]);
				exit( EXIT_FAILURE );
			}
			focal[i].active = 0;
		}
	}
	/* 1.3. Read configuration from arguments */
	else {
		/* 1.3.1. Check minimum number of arguments */
		if (argc<6) {
			fprintf(stderr, "-- Error in arguments: not enough arguments when reading configuration from the command line\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}

		/* 1.3.2. Surface and maximum number of iterations */
		rows = atoi( argv[1] );
		columns = atoi( argv[2] );
		max_iter = atoi( argv[3] );

		surface = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );
		surfaceCopy = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );

		/* 1.3.3. Teams information */
		num_teams = atoi( argv[4] );
		teams = (Team *)malloc( sizeof(Team) * (size_t)num_teams );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		if ( argc < num_teams*3 + 5 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_teams; i++ ) {
			teams[i].x = atoi( argv[5+i*3] );
			teams[i].y = atoi( argv[6+i*3] );
			teams[i].type = atoi( argv[7+i*3] );
		}

		/* 1.3.4. Focal points information */
		int focal_args = 5 + i*3;
		if ( argc < focal_args+1 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for the number of focal points\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
		num_focal = atoi( argv[focal_args] );
		focal = (FocalPoint *)malloc( sizeof(FocalPoint) * (size_t)num_focal );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		if ( argc < focal_args + 1 + num_focal*4 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_focal; i++ ) {
			focal[i].x = atoi( argv[focal_args+i*4+1] );
			focal[i].y = atoi( argv[focal_args+i*4+2] );
			focal[i].start = atoi( argv[focal_args+i*4+3] );
			focal[i].heat = atoi( argv[focal_args+i*4+4] );
			focal[i].active = 0;
		}

		/* 1.3.5. Sanity check: No extra arguments at the end of line */
		if ( argc > focal_args+i*4+1 ) {
			fprintf(stderr,"-- Error in arguments: extra arguments at the end of the command line\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
	}


#ifdef DEBUG
	/* 1.4. Print arguments */
	printf("Arguments, Rows: %d, Columns: %d, max_iter: %d\n", rows, columns, max_iter);
	printf("Arguments, Teams: %d, Focal points: %d\n", num_teams, num_focal );
	for( i=0; i<num_teams; i++ ) {
		printf("\tTeam %d, position (%d,%d), type: %d\n", i, teams[i].x, teams[i].y, teams[i].type );
	}
	for( i=0; i<num_focal; i++ ) {
		printf("\tFocal_point %d, position (%d,%d), start time: %d, temperature: %d\n", i, 
		focal[i].x,
		focal[i].y,
		focal[i].start,
		focal[i].heat );
	}
#endif // DEBUG

	/* 2. Select GPU and start global timer */
	hipSetDevice(0);
	hipDeviceSynchronize();
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */
	int const THREADS_PER_BLOCK = 128;
	hipError_t error;
	size_t surface_size_bytes = sizeof(float) * (rows * columns);
	int surface_size = rows*columns;

	//Allocate memory for the surface and surfaceCopy matrices in the GPU
	float *gpu_surface; 
	float *gpu_surfaceCopy;
	FocalPoint *gpu_focal;
	error = hipMalloc((void**)&gpu_surface, surface_size_bytes );
	if(error != hipSuccess){
		printf("Error allocating memory in the GPU\n");
	}
	error = hipMalloc((void**)&gpu_surfaceCopy, surface_size_bytes );
	if(error != hipSuccess){
		printf("Error allocating memory in the GPU\n");
	}
	error = hipMalloc((void**)&gpu_focal, sizeof(FocalPoint) * num_focal );
	if(error != hipSuccess){
		printf("Error allocating memory in the GPU\n");
	}

	//Determine how many blocks to use based on the number of elements and the number of threads
	//we want in each block
	int num_blocks = surface_size/THREADS_PER_BLOCK;
	if(surface_size%THREADS_PER_BLOCK != 0){
		num_blocks++;
	}

	/* 3. Initialize surface */
	
	for( i=0; i<rows; i++ )
		for( j=0; j<columns; j++ )
			accessMat( surface, i, j ) = 0.0;
	

	/* 4. Simulation */
	int iter;
	int flag_stability = 0;
	int first_activation = 0;
	for( iter=0; iter<max_iter && ! flag_stability; iter++ ) {

		/* 4.1. Activate focal points */
		int num_deactivated = 0;
		for( i=0; i<num_focal; i++ ) {
			if ( focal[i].start == iter ) {
				focal[i].active = 1;
				if ( ! first_activation ) first_activation = 1;
			}
			// Count focal points already deactivated by a team
			if ( focal[i].active == 2 ) num_deactivated++;
		}

		//Copy focal to the gpu
		error = hipMemcpy(gpu_focal, focal, sizeof(FocalPoint) * num_focal, hipMemcpyHostToDevice);
		if(error != hipSuccess){
			printf("Focal copy error\n");
		}

		/* 4.2. Propagate heat (10 steps per each team movement) */
		float global_residual = 0.0f;
		int step;

		//Send data from both surface and surfaceCopy to the GPU
		error = hipMemcpy(gpu_surface, surface, surface_size_bytes, hipMemcpyHostToDevice);
		if(error != hipSuccess){
			printf("1An error occurred while transfering data to the GPU\n");
			printf("Error: %s\n", hipGetErrorString( error ));
			printf("SurfaceSizeBytes: %d", surface_size_bytes);
		}
		error = hipMemcpy(gpu_surfaceCopy, surfaceCopy, surface_size_bytes, hipMemcpyHostToDevice);
		if(error != hipSuccess){
			printf("2An error occurred while transfering data to the GPU\n");
			printf("Error: %s\n", hipGetErrorString( error ));
		}

		for( step=0; step<10; step++ )	{			
			/* 4.2.1. Update heat on active focal points */
			update_focal<<<num_blocks, THREADS_PER_BLOCK>>>(gpu_surface, gpu_focal, num_focal, columns);

			hipMemcpy(gpu_surfaceCopy, gpu_surface, surface_size_bytes, hipMemcpyDeviceToDevice);
			//Swap pointers
			/*
			float* temp = gpu_surface;
			gpu_surface = gpu_surfaceCopy;
			gpu_surfaceCopy = temp;
			*/

			/* 4.2.3. Update surface values (skip borders) */
			update_heat<<<num_blocks, THREADS_PER_BLOCK>>>(gpu_surface, gpu_surfaceCopy, surface_size, rows, columns);
 
			int num_blocks_reduction;
			/* 4.2.4. Compute the maximum residual difference (absolute value) */
			if(num_deactivated == num_focal){
				if(step == 0){	
					compute_residual<<<num_blocks, THREADS_PER_BLOCK>>>(gpu_surface, gpu_surfaceCopy, surface_size, rows, columns);
					for(int reduction_size = surface_size; reduction_size > 1; reduction_size/=2){
						num_blocks_reduction = (reduction_size/2)/THREADS_PER_BLOCK;
						if(num_blocks_reduction==0){
							num_blocks_reduction++;
						}
						reduce_max<<<num_blocks_reduction, THREADS_PER_BLOCK>>>(gpu_surfaceCopy, reduction_size);
					}
					error = hipMemcpy(&global_residual, gpu_surfaceCopy, sizeof(float), hipMemcpyDeviceToHost);
				}
			}
		}

		//Return the updated data to the CPU
		error = hipMemcpy(surface, gpu_surface, surface_size_bytes, hipMemcpyDeviceToHost);
		if(error != hipSuccess){
			printf("3An error occurred while transfering data to the CPU\n");
			printf("Error: %s\n", hipGetErrorString( error ));
		}

		/* If the global residual is lower than THRESHOLD, we have reached enough stability, stop simulation at the end of this iteration */
		if( num_deactivated == num_focal && global_residual < THRESHOLD ) flag_stability = 1;

		/* 4.3. Move teams */
		for( t=0; t<num_teams; t++ ) {
			/* 4.3.1. Choose nearest focal point */
			float distance = FLT_MAX;
			int target = -1;
			for( j=0; j<num_focal; j++ ) {
				if ( focal[j].active != 1 ) continue; // Skip non-active focal points
				float dx = focal[j].x - teams[t].x;
				float dy = focal[j].y - teams[t].y;
				float local_distance = sqrtf( dx*dx + dy*dy );
				if ( local_distance < distance ) {
					distance = local_distance;
					target = j;
				}
			}
			/* 4.3.2. Annotate target for the next stage */
			teams[t].target = target;

			/* 4.3.3. No active focal point to choose, no movement */
			if ( target == -1 ) continue; 

			/* 4.3.4. Move in the focal point direction */
			if ( teams[t].type == 1 ) { 
				// Type 1: Can move in diagonal
				if ( focal[target].x < teams[t].x ) teams[t].x--;
				if ( focal[target].x > teams[t].x ) teams[t].x++;
				if ( focal[target].y < teams[t].y ) teams[t].y--;
				if ( focal[target].y > teams[t].y ) teams[t].y++;
			}
			else if ( teams[t].type == 2 ) { 
				// Type 2: First in horizontal direction, then in vertical direction
				if ( focal[target].y < teams[t].y ) teams[t].y--;
				else if ( focal[target].y > teams[t].y ) teams[t].y++;
				else if ( focal[target].x < teams[t].x ) teams[t].x--;
				else if ( focal[target].x > teams[t].x ) teams[t].x++;
			}
			else {
				// Type 3: First in vertical direction, then in horizontal direction
				if ( focal[target].x < teams[t].x ) teams[t].x--;
				else if ( focal[target].x > teams[t].x ) teams[t].x++;
				else if ( focal[target].y < teams[t].y ) teams[t].y--;
				else if ( focal[target].y > teams[t].y ) teams[t].y++;
			}
		}

		/* 4.4. Team actions */
		for( t=0; t<num_teams; t++ ) {
			/* 4.4.1. Deactivate the target focal point when it is reached */
			int target = teams[t].target;
			if ( target != -1 && focal[target].x == teams[t].x && focal[target].y == teams[t].y 
				&& focal[target].active == 1 )
				focal[target].active = 2;

			/* 4.4.2. Reduce heat in a circle around the team */
			int radius;
			// Influence area of fixed radius depending on type
			if ( teams[t].type == 1 ) radius = RADIUS_TYPE_1;
			else radius = RADIUS_TYPE_2_3;
			for( i=teams[t].x-radius; i<=teams[t].x+radius; i++ ) {
				for( j=teams[t].y-radius; j<=teams[t].y+radius; j++ ) {
					if ( i<1 || i>=rows-1 || j<1 || j>=columns-1 ) continue; // Out of the heated surface
					float dx = teams[t].x - i;
					float dy = teams[t].y - j;
					float distance = sqrtf( dx*dx + dy*dy );
					if ( distance <= radius ) {
						accessMat( surface, i, j ) = accessMat( surface, i, j ) * ( 1 - 0.25 ); // Team efficiency factor
					}
				}
			}
		}

#ifdef DEBUG
		/* 4.5. DEBUG: Print the current state of the simulation at the end of each iteration */
		print_status( iter, rows, columns, surface, num_teams, teams, num_focal, focal, global_residual );
#endif // DEBUG
	}
	
/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	/* 5. Stop global time */
	hipDeviceSynchronize();
	ttotal = cp_Wtime() - ttotal;

	/* 6. Output for leaderboard */
	printf("\n");
	/* 6.1. Total computation time */
	printf("Time: %lf\n", ttotal );
	/* 6.2. Results: Number of iterations, position of teams, residual heat on the focal points */
	printf("Result: %d", iter);
	/*
	for (i=0; i<num_teams; i++)
		printf(" %d %d", teams[i].x, teams[i].y );
	*/
	for (i=0; i<num_focal; i++)
		printf(" %.6f", accessMat( surface, focal[i].x, focal[i].y ) );
	printf("\n");

	/* 7. Free resources */	
	free( teams );
	free( focal );
	free( surface );
	free( surfaceCopy );

	/* 8. End */
	return 0;
}
