#include "hip/hip_runtime.h"
// User: g305@62.42.14.243 
// ExecutionRequest[P:'despacito.cu',P:1,T:1,args:'',q:'cudalb'] 
// May 15 2019 21:49:36
#include "cputils.h" // Added by tablon
/*
 * Simplified simulation of fire extinguishing
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2018/2019
 *
 * v1.4
 *
 * (c) 2019 Arturo Gonzalez Escribano
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <cputils.h>

#define RADIUS_TYPE_1		3
#define RADIUS_TYPE_2_3		9
#define THRESHOLD	0.1f

/* Structure to store data of an extinguishing team */
typedef struct {
	int x,y;
	int type;
	int target;
} Team;

/* Structure to store data of a fire focal point */
typedef struct {
	int x,y;
	int start;
	int heat;
	int active; // States: 0 Not yet activated; 1 Active; 2 Deactivated by a team
} FocalPoint;

/* Macro function to simplify accessing with two coordinates to a flattened array */
#define accessMat( arr, exp1, exp2 )	arr[ (exp1) * columns + (exp2) ]


/*Kernels CUDA*************************************************************************************************/
__global__ void  icicializa(float *array){
	array[ blockIdx.x *blockDim.x +  threadIdx.x ] = 0;
}



__global__ void  calculaGlobal(float *arrayDevice, const float *surface, const float *surfaceCopy){
	//arrayDevice[ blockIdx.x *blockDim.x +  threadIdx.x ] = 0;
	int gid = blockIdx.x *blockDim.x +  threadIdx.x;
	//arrayDevice[ gid ]; //= /*fabs(*/ surface[gid]; //- surfaceCopy[blockIdx.x * blockDim.x + threadIdx.x]);
	arrayDevice[ gid ]=fabs(surface[gid]-surfaceCopy[gid]);
	//printf("%lf\n", arrayDevice[ gid ]);

}

__device__ void warpReduce(volatile float* sdata, int tid) {
	sdata[tid]=sdata[tid] > sdata[tid + 32] ? sdata[tid]: sdata[tid + 32];
	sdata[tid]=sdata[tid] > sdata[tid + 16] ? sdata[tid]: sdata[tid + 16];
	sdata[tid]=sdata[tid] > sdata[tid + 8] ? sdata[tid]: sdata[tid + 8];
	sdata[tid]=sdata[tid] > sdata[tid + 4] ? sdata[tid]: sdata[tid + 4];
	sdata[tid]=sdata[tid] > sdata[tid + 2] ? sdata[tid]: sdata[tid + 2];
	sdata[tid]=sdata[tid] > sdata[tid + 1] ? sdata[tid]: sdata[tid + 1];
}


__global__ void reduce0(float *g_idata, float *g_odata, int size) {
	extern __shared__ float sdata[];
	// each thread loads one element from global to shared mem
	/*unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i];
	__syncthreads();*/

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	sdata[tid] = g_idata[i] > g_idata[i+blockDim.x] ? g_idata[i] : g_idata[i+blockDim.x];
	__syncthreads();

	// do reduction in shared mem
	//for(unsigned int s=1; s < blockDim.x; s *= 2) {
		//if (tid % (2*s) == 0) {
		for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
			if (tid < s)
				if(sdata[tid]<sdata[tid + s]){
					sdata[tid] = sdata[tid + s];
				}
				// Reducción suplementaria si hay un elemento desparejado
				if ( size%2 != 0 && i == 0 ){ // ¿Quién se encarga de hacer la reducción?
					if(g_idata[ i ] < g_idata[ size-1 ]){ // ¿Dónde está el elemento desparejado?
						g_odata[ i ] = g_idata[ size-1 ];
					}
				}
			__syncthreads();
		}
		// Reducción suplementaria si hay un elemento desparejado
		if ( size%2 != 0 && i == 0 ){ // ¿Quién se encarga de hacer la reducción?
			if(g_idata[ i ] < g_idata[ size-1 ]){ // ¿Dónde está el elemento desparejado?
				g_odata[ i ] = g_idata[ size-1 ];
			}
		}
		__syncthreads();
		if (tid <= 32) warpReduce(sdata, tid);


		__syncthreads();
	//}
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}



__global__ void reduce1(float *g_idata, float *g_odata, int size) {
	extern __shared__ float sdata[];
	// each thread loads one element from global to shared mem
	/*unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i];
	__syncthreads();*/

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	sdata[tid] = g_idata[i] > g_idata[i+blockDim.x] ? g_idata[i] : g_idata[i+blockDim.x];
	__syncthreads();

	// do reduction in shared mem
	//for(unsigned int s=1; s < blockDim.x; s *= 2) {
		//if (tid % (2*s) == 0) {
		for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
			if (tid < s)
				if(sdata[tid]<sdata[tid + s]){
					sdata[tid] = sdata[tid + s];
				}
				// Reducción suplementaria si hay un elemento desparejado
				if ( size%2 != 0 && i == 0 ){ // ¿Quién se encarga de hacer la reducción?
					if(g_idata[ i ] < g_idata[ size-1 ]){ // ¿Dónde está el elemento desparejado?
						g_odata[ i ] = g_idata[ size-1 ];
					}
				}
			__syncthreads();
		}
		// Reducción suplementaria si hay un elemento desparejado
		if ( size%2 != 0 && i == 0 ){ // ¿Quién se encarga de hacer la reducción?
			if(g_idata[ i ] < g_idata[ size-1 ]){ // ¿Dónde está el elemento desparejado?
				g_odata[ i ] = g_idata[ size-1 ];
			}
		}
		__syncthreads();
		if (tid <= 32) warpReduce(sdata, tid);


		__syncthreads();
	//}
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void actualiza(float *actualizar, const float *surface, const float *surfaceCopy, int columns, int rows) {
	int gid = blockIdx.x *blockDim.x + threadIdx.x;
	/*int col= gid/columns;
	int fila=(blockIdx.x*blockDim.x/columns)+(threadIdx.x/columns);*/

	//int col= threadIdx.x;
	//int fila= blockIdx.x;

	if( blockIdx.x>0 && blockIdx.x<rows-1 && threadIdx.x>0 && threadIdx.x<columns-1){
		accessMat(actualizar,blockIdx.x,threadIdx.x)= (accessMat( surfaceCopy, blockIdx.x-1, threadIdx.x ) +
																										accessMat( surfaceCopy, blockIdx.x+1, threadIdx.x ) +
																										accessMat( surfaceCopy, blockIdx.x, threadIdx.x-1 ) +
																										accessMat( surfaceCopy, blockIdx.x, threadIdx.x+1 ) ) / 4;
																							//(surfaceCopy[gid-blockDim.x]+surfaceCopy[gid+blockDim.x]+surfaceCopy[gid-1]+surfaceCopy[gid+1])/4;
		//if(gid==59){printf("columna %d, fila %d\n",col, fila);}
	}else{
		actualizar[gid]=0.0f;
	}
}

//__global__ void actualiza2(float *surface, const float *surfaceCopy2, const float *surfaceCopy) {
//}


/****************************************************************************************************************/







/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s <config_file> | <command_line_args>\n", program_name );
	fprintf(stderr,"\t<config_file> ::= -f <file_name>\n");
	fprintf(stderr,"\t<command_line_args> ::= <rows> <columns> <maxIter> <numTeams> [ <teamX> <teamY> <teamType> ... ] <numFocalPoints> [ <focalX> <focalY> <focalStart> <focalTemperature> ... ]\n");
	fprintf(stderr,"\n");
}

#ifdef DEBUG
/*
 * Function: Print the current state of the simulation
 */
void print_status( int iteration, int rows, int columns, float *surface, int num_teams, Team *teams, int num_focal, FocalPoint *focal, float global_residual ) {
	/*
	 * You don't need to optimize this function, it is only for pretty printing and debugging purposes.
	 * It is not compiled in the production versions of the program.
	 * Thus, it is never used when measuring times in the leaderboard
	 */
	int i,j;

	printf("Iteration: %d\n", iteration );
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	for( i=0; i<rows; i++ ) {
		printf("|");
		for( j=0; j<columns; j++ ) {
			char symbol;
			if ( accessMat( surface, i, j ) >= 1000 ) symbol = '*';
			else if ( accessMat( surface, i, j ) >= 100 ) symbol = '0' + (int)(accessMat( surface, i, j )/100);
			else if ( accessMat( surface, i, j ) >= 50 ) symbol = '+';
			else if ( accessMat( surface, i, j ) >= 25 ) symbol = '.';
			else symbol = '0';

			int t;
			int flag_team = 0;
			for( t=0; t<num_teams; t++ )
				if ( teams[t].x == i && teams[t].y == j ) { flag_team = 1; break; }
			if ( flag_team ) printf("[%c]", symbol );
			else {
				int f;
				int flag_focal = 0;
				for( f=0; f<num_focal; f++ )
					if ( focal[f].x == i && focal[f].y == j && focal[f].active == 1 ) { flag_focal = 1; break; }
				if ( flag_focal ) printf("(%c)", symbol );
				else printf(" %c ", symbol );
			}
		}
		printf("|\n");
	}
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	printf("Global residual: %f\n\n", global_residual);
}
#endif

/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	int i,j,t;

	// Simulation data
	int rows, columns, max_iter;
	float *surface, *surfaceCopy;
	int num_teams, num_focal;
	Team *teams;
	FocalPoint *focal;

	/* 1. Read simulation arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc<2) {
		fprintf(stderr,"-- Error in arguments: No arguments\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	int read_from_file = ! strcmp( argv[1], "-f" );
	/* 1.2. Read configuration from file */
	if ( read_from_file ) {
		/* 1.2.1. Open file */
		if (argc<3) {
			fprintf(stderr,"-- Error in arguments: file-name argument missing\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
		FILE *args = cp_abrir_fichero( argv[2] );
		if ( args == NULL ) {
			fprintf(stderr,"-- Error in file: not found: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}

		/* 1.2.2. Read surface and maximum number of iterations */
		int ok;
		ok = fscanf(args, "%d %d %d", &rows, &columns, &max_iter);
		if ( ok != 3 ) {
			fprintf(stderr,"-- Error in file: reading rows, columns, max_iter from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}

		surface = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );
		surfaceCopy = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );

		if ( surface == NULL || surfaceCopy == NULL ) {
			fprintf(stderr,"-- Error allocating: surface structures\n");
			exit( EXIT_FAILURE );
		}

		/* 1.2.3. Teams information */
		ok = fscanf(args, "%d", &num_teams );
		if ( ok != 1 ) {
			fprintf(stderr,"-- Error file, reading num_teams from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}
		teams = (Team *)malloc( sizeof(Team) * (size_t)num_teams );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_teams; i++ ) {
			ok = fscanf(args, "%d %d %d", &teams[i].x, &teams[i].y, &teams[i].type);
			if ( ok != 3 ) {
				fprintf(stderr,"-- Error in file: reading team %d from file: %s\n", i, argv[1]);
				exit( EXIT_FAILURE );
			}
		}

		/* 1.2.4. Focal points information */
		ok = fscanf(args, "%d", &num_focal );
		if ( ok != 1 ) {
			fprintf(stderr,"-- Error in file: reading num_focal from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}
		focal = (FocalPoint *)malloc( sizeof(FocalPoint) * (size_t)num_focal );
		if ( focal == NULL ) {
			fprintf(stderr,"-- Error allocating: %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_focal; i++ ) {
			ok = fscanf(args, "%d %d %d %d", &focal[i].x, &focal[i].y, &focal[i].start, &focal[i].heat);
			if ( ok != 4 ) {
				fprintf(stderr,"-- Error in file: reading focal point %d from file: %s\n", i, argv[1]);
				exit( EXIT_FAILURE );
			}
			focal[i].active = 0;
		}
	}
	/* 1.3. Read configuration from arguments */
	else {
		/* 1.3.1. Check minimum number of arguments */
		if (argc<6) {
			fprintf(stderr, "-- Error in arguments: not enough arguments when reading configuration from the command line\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}

		/* 1.3.2. Surface and maximum number of iterations */
		rows = atoi( argv[1] );
		columns = atoi( argv[2] );
		max_iter = atoi( argv[3] );

		surface = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );
		surfaceCopy = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );

		/* 1.3.3. Teams information */
		num_teams = atoi( argv[4] );
		teams = (Team *)malloc( sizeof(Team) * (size_t)num_teams );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		if ( argc < num_teams*3 + 5 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_teams; i++ ) {
			teams[i].x = atoi( argv[5+i*3] );
			teams[i].y = atoi( argv[6+i*3] );
			teams[i].type = atoi( argv[7+i*3] );
		}

		/* 1.3.4. Focal points information */
		int focal_args = 5 + i*3;
		if ( argc < focal_args+1 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for the number of focal points\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
		num_focal = atoi( argv[focal_args] );
		focal = (FocalPoint *)malloc( sizeof(FocalPoint) * (size_t)num_focal );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		if ( argc < focal_args + 1 + num_focal*4 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_focal; i++ ) {
			focal[i].x = atoi( argv[focal_args+i*4+1] );
			focal[i].y = atoi( argv[focal_args+i*4+2] );
			focal[i].start = atoi( argv[focal_args+i*4+3] );
			focal[i].heat = atoi( argv[focal_args+i*4+4] );
			focal[i].active = 0;
		}

		/* 1.3.5. Sanity check: No extra arguments at the end of line */
		if ( argc > focal_args+i*4+1 ) {
			fprintf(stderr,"-- Error in arguments: extra arguments at the end of the command line\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
	}


#ifdef DEBUG
	/* 1.4. Print arguments */
	printf("Arguments, Rows: %d, Columns: %d, max_iter: %d\n", rows, columns, max_iter);
	printf("Arguments, Teams: %d, Focal points: %d\n", num_teams, num_focal );
	for( i=0; i<num_teams; i++ ) {
		printf("\tTeam %d, position (%d,%d), type: %d\n", i, teams[i].x, teams[i].y, teams[i].type );
	}
	for( i=0; i<num_focal; i++ ) {
		printf("\tFocal_point %d, position (%d,%d), start time: %d, temperature: %d\n", i,
		focal[i].x,
		focal[i].y,
		focal[i].start,
		focal[i].heat );
	}
#endif // DEBUG

	/* 2. Select GPU and start global timer */
	hipSetDevice(0);
	hipDeviceSynchronize();
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
  *********************************************************1,94 s cuda/ 1m 21s cudalb
 */
 #define BLOCK_SIZE 128
 #define CUDA_CHECK()  { \
	 hipError_t check = hipGetLastError(); \
	 if ( check != hipSuccess ) { 		\
		 printf("Error.... %s \n", hipGetErrorString( check ) ); \
		 exit( EXIT_FAILURE ); \
	 } }

	 unsigned int grid_size = rows*columns / BLOCK_SIZE + (rows*columns % BLOCK_SIZE ? 1 : 0);
	 unsigned int block_size = BLOCK_SIZE;
	 unsigned int aux_grid;

 float *arrayDevice;
 hipMalloc( (void**) &arrayDevice, sizeof(float) * (size_t)rows * (size_t)columns);
 float *arrayCopyDevice;
 hipMalloc( (void**) &arrayCopyDevice, sizeof(float) * (size_t)rows * (size_t)columns);

 float *globalSurface = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );
 float *globalDevice;
 hipMalloc( (void**) &globalDevice, sizeof(float) * (size_t)rows * (size_t)columns);

	float *actualizaDevice;
	hipMalloc( (void**) &actualizaDevice, sizeof(float) * (size_t)rows * (size_t)columns);

	//CUDA_CHECK();

	/* 3. Initialize surface */
	//icicializa<<<grid_size, block_size>>>( arrayDevice );
	//hipMemcpy(surface,arrayDevice, sizeof(float) * (size_t)rows * (size_t)columns, hipMemcpyDeviceToHost);

	for( i=0; i<rows; i++ )
		for( j=0; j<columns; j++ )
			/*printf("%lf\n", accessMat( surface, i, j ));*/accessMat( surface, i, j ) = 0.0;

	/* 4. Simulation */
	int iter;
	int flag_stability = 0;
	int first_activation = 0;
	for( iter=0; iter<max_iter && ! flag_stability; iter++ ) {

		/* 4.1. Activate focal points */
		int num_deactivated = 0;
		for( i=0; i<num_focal; i++ ) {
			if ( focal[i].start == iter ) {
				focal[i].active = 1;
				if ( ! first_activation ) first_activation = 1;
			}
			// Count focal points already deactivated by a team
			if ( focal[i].active == 2 ) num_deactivated++;
		}

		if(first_activation){
		/* 4.2. Propagate heat (10 steps per each team movement) */
		float global_residual = 0.0f;
		int step;
		for( step=0; step<10; step++ )	{
			/* 4.2.1. Update heat on active focal points */
			for( i=0; i<num_focal; i++ ) {
				if ( focal[i].active != 1 ) continue;
				//int x = focal[i].x;
				//int y = focal[i].y;
				accessMat( surface, focal[i].x, focal[i].y ) = focal[i].heat;
			}

			//Copia optimizada
			 float *aux=surface;
			 surface=surfaceCopy;
			 surfaceCopy=aux;

			/* 4.2.2. Copy values of the surface in ancillary structure (Skip borders) */
			/*for( i=1; i<rows-1; i++ )
				for( j=1; j<columns-1; j++ )
					accessMat( surfaceCopy, i, j ) = accessMat( surface, i, j );*/

			//hipMemcpy(globalDevice, surface, sizeof(float) * (size_t)rows * (size_t)columns,hipMemcpyHostToDevice);
			//hipMemcpy(arrayDevice, surface, sizeof(float) * (size_t)rows * (size_t)columns,hipMemcpyHostToDevice);
			/*hipMemcpy(arrayCopyDevice, surfaceCopy, sizeof(float) * (size_t)rows * (size_t)columns,hipMemcpyHostToDevice);
			actualiza<<<columns, rows>>>( globalDevice, arrayDevice, arrayCopyDevice, columns, rows);
			hipMemcpy(surface,globalDevice, sizeof(float) * (size_t)rows * (size_t)columns, hipMemcpyDeviceToHost);

			//CUDA_CHECK();
			/* 4.2.3. Update surface values (skip borders) */
			for( i=1; i<rows-1; i++ )
				for( j=1; j<columns-1; j++ )
					accessMat( surface, i, j ) = (
						accessMat( surfaceCopy, i-1, j ) +
						accessMat( aux, i+1, j ) +
						accessMat( aux, i, j-1 ) +
						accessMat( surfaceCopy, i, j+1 ) ) / 4;

			/* 4.2.4. Compute the maximum residual difference (absolute value) */

			if(step==0){//Probar luego con <1
				//Trabajar en los kernel con variables de dispositivo
	/*			hipMemcpy(arrayDevice, surface, sizeof(float) * (size_t)rows * (size_t)columns,hipMemcpyHostToDevice);
				//hipMemcpy(globalDevice, surface, sizeof(float) * (size_t)rows * (size_t)columns,hipMemcpyHostToDevice);
				hipMemcpy(arrayCopyDevice, surfaceCopy, sizeof(float) * (size_t)rows * (size_t)columns,hipMemcpyHostToDevice);
				//Calcular global en cada posicion
				calculaGlobal<<<grid_size, block_size>>>( arrayDevice,globalDevice, arrayCopyDevice);
				hipMemcpy(globalSurface,arrayDevice, sizeof(float) * (size_t)rows * (size_t)columns, hipMemcpyDeviceToHost);
				CUDA_CHECK();*/
				//printf("Calculos hechos\n");

				/*for( i=1; i<rows-1; i++ )
					for( j=1; j<columns-1; j++ )
						accessMat( arrayDevice, i, j ) = fabs( accessMat( surface, i, j ) - accessMat( surfaceCopy, i, j ) );*/

				//for (int redSize = rows*columns; redSize>1; redSize /= 2) {
					// Reducción por niveles en la GPU
					//reduceGlobal<<< grid_size, block_size >>>( arrayDevice, arrayDevice, redSize );
					/*reduce0<<< grid_size, block_size >>>( arrayDevice, arrayDevice );
					CUDA_CHECK();
					// ¿Es necesario sincronizar explícitamente los kernels entre niveles?
				//}
				hipMemcpy(&global_residual, arrayDevice, sizeof(float), hipMemcpyDeviceToHost);
				CUDA_CHECK();
*/
			/*float abc=0.0f;
			printf("%lf\n", globalSurface[4] );
			for(int a=0;a<sizeof(globalSurface);a++){
					if(globalSurface[a]>abc){
						abc=globalSurface[a];
					}
			// }*/
			//printf("Valor calculado con kernel %lf\n", abc );
				for( i=1; i<rows-1; i++ )
					for( j=1; j<columns-1; j++ )
						if ( fabs( accessMat( surface, i, j ) - accessMat( surfaceCopy, i, j ) ) > global_residual ) {
							global_residual = fabs( accessMat( surface, i, j ) - accessMat( surfaceCopy, i, j ) );
						}
			}
		}
		/* If the global residual is lower than THRESHOLD, we have reached enough stability, stop simulation at the end of this iteration */
		if( !(num_deactivated == num_focal && global_residual < THRESHOLD) ){ //flag_stability = 1;

		/* 4.3. Move teams */
		for( t=0; t<num_teams; t++ ) {
			/* 4.3.1. Choose nearest focal point */
			float distance = FLT_MAX;
			int target = -1;
			for( j=0; j<num_focal; j++ ) {
				if ( focal[j].active != 1 ) continue; // Skip non-active focal points
				float dx = focal[j].x - teams[t].x;
				float dy = focal[j].y - teams[t].y;
				float local_distance = sqrtf( dx*dx + dy*dy );
				if ( local_distance < distance ) {
					distance = local_distance;
					target = j;
				}
			}
			/* 4.3.2. Annotate target for the next stage */
			teams[t].target = target;

			/* 4.3.3. No active focal point to choose, no movement */
			if ( target == -1 ) continue;

			/* 4.3.4. Move in the focal point direction */
			if ( teams[t].type == 1 ) {
				// Type 1: Can move in diagonal
				if ( focal[target].x < teams[t].x ) teams[t].x--;
				if ( focal[target].x > teams[t].x ) teams[t].x++;
				if ( focal[target].y < teams[t].y ) teams[t].y--;
				if ( focal[target].y > teams[t].y ) teams[t].y++;
			}
			else if ( teams[t].type == 2 ) {
				// Type 2: First in horizontal direction, then in vertical direction
				if ( focal[target].y < teams[t].y ) teams[t].y--;
				else if ( focal[target].y > teams[t].y ) teams[t].y++;
				else if ( focal[target].x < teams[t].x ) teams[t].x--;
				else if ( focal[target].x > teams[t].x ) teams[t].x++;
			}
			else {
				// Type 3: First in vertical direction, then in horizontal direction
				if ( focal[target].x < teams[t].x ) teams[t].x--;
				else if ( focal[target].x > teams[t].x ) teams[t].x++;
				else if ( focal[target].y < teams[t].y ) teams[t].y--;
				else if ( focal[target].y > teams[t].y ) teams[t].y++;
			}
		}

		/* 4.4. Team actions */
		for( t=0; t<num_teams; t++ ) {
			/* 4.4.1. Deactivate the target focal point when it is reached */
			int target = teams[t].target;
			if ( target != -1 && focal[target].x == teams[t].x && focal[target].y == teams[t].y
				&& focal[target].active == 1 )
				focal[target].active = 2;

			/* 4.4.2. Reduce heat in a circle around the team */
			int radius;
			// Influence area of fixed radius depending on type
			if ( teams[t].type == 1 ) radius = RADIUS_TYPE_1;
			else radius = RADIUS_TYPE_2_3;
			for( i=teams[t].x-radius; i<=teams[t].x+radius; i++ ) {
				for( j=teams[t].y-radius; j<=teams[t].y+radius; j++ ) {
					if ( i<1 || i>=rows-1 || j<1 || j>=columns-1 ) continue; // Out of the heated surface
					float dx = teams[t].x - i;
					float dy = teams[t].y - j;
					float distance = sqrtf( dx*dx + dy*dy );
					if ( distance <= radius ) {
						accessMat( surface, i, j ) = accessMat( surface, i, j ) *  0.75; // Team efficiency factor
					}
				}
			}
		}
}else{
	flag_stability = 1;
	/* 4.4. Team actions */
	for( t=0; t<num_teams; t++ ) {
		/* 4.4.1. Deactivate the target focal point when it is reached */
		int target = teams[t].target;
		if ( target != -1 && focal[target].x == teams[t].x && focal[target].y == teams[t].y
			&& focal[target].active == 1 )
			focal[target].active = 2;

		/* 4.4.2. Reduce heat in a circle around the team */
		int radius;
		// Influence area of fixed radius depending on type
		if ( teams[t].type == 1 ) radius = RADIUS_TYPE_1;
		else radius = RADIUS_TYPE_2_3;
		for( i=teams[t].x-radius; i<=teams[t].x+radius; i++ ) {
			for( j=teams[t].y-radius; j<=teams[t].y+radius; j++ ) {
				if ( i<1 || i>=rows-1 || j<1 || j>=columns-1 ) continue; // Out of the heated surface
				float dx = teams[t].x - i;
				float dy = teams[t].y - j;
				float distance = sqrtf( dx*dx + dy*dy );
				if ( distance <= radius ) {
					accessMat( surface, i, j ) = accessMat( surface, i, j ) *  0.75; // Team efficiency factor
				}
			}
		}
	}
}
}
#ifdef DEBUG
		/* 4.5. DEBUG: Print the current state of the simulation at the end of each iteration */
		print_status( iter, rows, columns, surface, num_teams, teams, num_focal, focal, global_residual );
#endif // DEBUG
	}

/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	/* 5. Stop global time */
	hipDeviceSynchronize();
	ttotal = cp_Wtime() - ttotal;

	/* 6. Output for leaderboard */
	printf("\n");
	/* 6.1. Total computation time */
	printf("Time: %lf\n", ttotal );
	/* 6.2. Results: Number of iterations, position of teams, residual heat on the focal points */
	printf("Result: %d", iter);
	/*
	for (i=0; i<num_teams; i++)
		printf(" %d %d", teams[i].x, teams[i].y );
	*/
	for (i=0; i<num_focal; i++)
		printf(" %.6f", accessMat( surface, focal[i].x, focal[i].y ) );
	printf("\n");

	/* 7. Free resources */
	free( teams );
	free( focal );
	free( surface );
	free( surfaceCopy );

	/* 8. End */
	return 0;
}
