#include "hip/hip_runtime.h"
// User: g301@62.42.159.187 
// ExecutionRequest[P:'extinguishing.cu',P:1,T:1,args:'',q:'cudalb'] 
// May 16 2019 13:50:58
#include "cputils.h" // Added by tablon
/*
 * Simplified simulation of fire extinguishing
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2018/2019
 *
 * v1.4
 *
 * (c) 2019 Arturo Gonzalez Escribano
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <cputils.h>
#include <hip/hip_runtime.h>

#define RADIUS_TYPE_1		3
#define RADIUS_TYPE_2_3		9
#define THRESHOLD	0.1f

/* Structure to store data of an extinguishing team */
typedef struct {
	int x,y;
	int type;
	int target;
} Team;

/* Structure to store data of a fire focal point */
typedef struct {
	int x,y;
	int start;
	int heat;
	int active; // States: 0 Not yet activated; 1 Active; 2 Deactivated by a team
} FocalPoint;

/* Macro function to simplify accessing with two coordinates to a flattened array */
#define accessMat( arr, exp1, exp2 )	arr[ (exp1) * columns + (exp2) ]

/*
*	Kernels
*/
__global__ void inicializar(float *surface, int ITEMS)
{
    int position = ((blockIdx.y*gridDim.x)+blockIdx.x)*(blockDim.x*blockDim.y)+((threadIdx.y*blockDim.x)+threadIdx.x);
    if(position<ITEMS)
      surface[position]=0.0f;

}

__global__ void actualizar(float *dsurface, float *dsurfacecopy, int rows, int columns)
{
  int position = ((blockIdx.y*gridDim.x)+blockIdx.x)*(blockDim.x*blockDim.y)+((threadIdx.y*blockDim.x)+threadIdx.x);
  int ITEMS = rows*columns;
	if(position > columns && position < (ITEMS - columns) && ((position%columns) != 0) && ((position%columns) != (columns-1)))
	{
    dsurface[position] =
      (dsurfacecopy[position-columns] +
      dsurfacecopy[position+columns] +
      dsurfacecopy[position-1] +
      dsurfacecopy[position+1]) / 4;
	}
}
__global__ void calc_global_residual(float *diferencia, int ITEMS, int columns)
{
	int gid = ((blockIdx.y*gridDim.x)+blockIdx.x)*(blockDim.x*blockDim.y)+((threadIdx.y*blockDim.x)+threadIdx.x);
	// Condición para evitar hilos ociosos por no tener pares asignados
	if ( gid >= (ITEMS/2) /*|| gid <= columns || ((gid%columns) == 0) || ((gid%columns) == (columns-1))*/) return;

	// Reducción del elemento en la posición gid con su pareja
	if(diferencia[gid] < diferencia[(ITEMS/2)+gid]){
			diferencia[ gid ] = diferencia[(ITEMS/2)+gid];
		}
	// Reducción suplementaria si hay un elemento desparejado
	/***************************************************************/
	/* NO DESCOMENTES ESTO HASTA QUE NO SE TE PIDA EN EL ENUNCIADO */
	/***************************************************************/

	if ( ITEMS%2 != 0 && gid == columns+1 ){ // ¿Quién se encarga de hacer la reducción?
			 if(diferencia[gid] < diferencia[ITEMS-1]){
		 			diferencia[ gid ] = diferencia[ITEMS-1];// ¿Dónde está el elemento desparejado?
				}
	}
	if (ITEMS <= 3){
		if(diferencia[columns+1] < diferencia[columns+2]){
			 diferencia[ columns+1 ] = diferencia[columns+2];
		 }
		 if(diferencia[columns+1] < diferencia[columns+3]){
 			 diferencia[ columns+1 ] = diferencia[columns+3];
 		 }
 	}

	//if(diferencia[gid]>0) printf("reduccion de %d: %f, num elementos %d \n", gid, diferencia[gid], ITEMS);

}

__global__ void calc_diferencia(float *dsurface, float *dsurfacecopy, float *diferencia, int ITEMS)
{
	int gid = ((blockIdx.y*gridDim.x)+blockIdx.x)*(blockDim.x*blockDim.y)+((threadIdx.y*blockDim.x)+threadIdx.x);
	if (gid >= ITEMS) return;
	if(gid<ITEMS) diferencia[gid] = fabs(dsurface[gid] - dsurfacecopy[gid]);
	//if(diferencia[gid]>0) printf("diferencia de %d: %f \n", gid, diferencia[gid]);
}



__global__ void actualizaCalor(float *dsurface, int num_focal, FocalPoint *focalaux, int columns)
{
    int position = ((blockIdx.y*gridDim.x)+blockIdx.x)*(blockDim.x*blockDim.y)+((threadIdx.y*blockDim.x)+threadIdx.x);
		//puede ser el if menor o igual
			if((position<num_focal)&&(focalaux[position].active == 1)){
					int x = focalaux[position].x;
					int y = focalaux[position].y;
					accessMat( dsurface, x, y ) = focalaux[position].heat;
				}
    }





/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s <config_file> | <command_line_args>\n", program_name );
	fprintf(stderr,"\t<config_file> ::= -f <file_name>\n");
	fprintf(stderr,"\t<command_line_args> ::= <rows> <columns> <maxIter> <numTeams> [ <teamX> <teamY> <teamType> ... ] <numFocalPoints> [ <focalX> <focalY> <focalStart> <focalTemperature> ... ]\n");
	fprintf(stderr,"\n");
}

#ifdef DEBUG


/*
 * Function: Print the current state of the simulation
 */
void print_status( int iteration, int rows, int columns, float *surface, int num_teams, Team *teams, int num_focal, FocalPoint *focal, float global_residual ) {
	/*
	 * You don't need to optimize this function, it is only for pretty printing and debugging purposes.
	 * It is not compiled in the production versions of the program.
	 * Thus, it is never used when measuring times in the leaderboard
	 */
	int i,j;

	printf("Iteration: %d\n", iteration );
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	for( i=0; i<rows; i++ ) {
		printf("|");
		for( j=0; j<columns; j++ ) {
			char symbol;
			if ( accessMat( surface, i, j ) >= 1000 ) symbol = '*';
			else if ( accessMat( surface, i, j ) >= 100 ) symbol = '0' + (int)(accessMat( surface, i, j )/100);
			else if ( accessMat( surface, i, j ) >= 50 ) symbol = '+';
			else if ( accessMat( surface, i, j ) >= 25 ) symbol = '.';
			else symbol = '0';

			int t;
			int flag_team = 0;
			for( t=0; t<num_teams; t++ )
				if ( teams[t].x == i && teams[t].y == j ) { flag_team = 1; break; }
			if ( flag_team ) printf("[%c]", symbol );
			else {
				int f;
				int flag_focal = 0;
				for( f=0; f<num_focal; f++ )
					if ( focal[f].x == i && focal[f].y == j && focal[f].active == 1 ) { flag_focal = 1; break; }
				if ( flag_focal ) printf("(%c)", symbol );
				else printf(" %c ", symbol );
			}
		}
		printf("|\n");
	}
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	printf("Global residual: %f\n\n", global_residual);
}
#endif

/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	int i,j,t;

	// Simulation data
	int rows, columns, max_iter;
	float *surface, *surfaceCopy;
	int num_teams, num_focal;
	Team *teams;
	FocalPoint *focal;

	/* 1. Read simulation arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc<2) {
		fprintf(stderr,"-- Error in arguments: No arguments\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	int read_from_file = ! strcmp( argv[1], "-f" );
	/* 1.2. Read configuration from file */
	if ( read_from_file ) {
		/* 1.2.1. Open file */
		if (argc<3) {
			fprintf(stderr,"-- Error in arguments: file-name argument missing\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
		FILE *args = cp_abrir_fichero( argv[2] );
		if ( args == NULL ) {
			fprintf(stderr,"-- Error in file: not found: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}

		/* 1.2.2. Read surface and maximum number of iterations */
		int ok;
		ok = fscanf(args, "%d %d %d", &rows, &columns, &max_iter);
		if ( ok != 3 ) {
			fprintf(stderr,"-- Error in file: reading rows, columns, max_iter from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}

		surface = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );
		surfaceCopy = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );

		if ( surface == NULL || surfaceCopy == NULL ) {
			fprintf(stderr,"-- Error allocating: surface structures\n");
			exit( EXIT_FAILURE );
		}

		/* 1.2.3. Teams information */
		ok = fscanf(args, "%d", &num_teams );
		if ( ok != 1 ) {
			fprintf(stderr,"-- Error file, reading num_teams from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}
		teams = (Team *)malloc( sizeof(Team) * (size_t)num_teams );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_teams; i++ ) {
			ok = fscanf(args, "%d %d %d", &teams[i].x, &teams[i].y, &teams[i].type);
			if ( ok != 3 ) {
				fprintf(stderr,"-- Error in file: reading team %d from file: %s\n", i, argv[1]);
				exit( EXIT_FAILURE );
			}
		}

		/* 1.2.4. Focal points information */
		ok = fscanf(args, "%d", &num_focal );
		if ( ok != 1 ) {
			fprintf(stderr,"-- Error in file: reading num_focal from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}
		focal = (FocalPoint *)malloc( sizeof(FocalPoint) * (size_t)num_focal );
		if ( focal == NULL ) {
			fprintf(stderr,"-- Error allocating: %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_focal; i++ ) {
			ok = fscanf(args, "%d %d %d %d", &focal[i].x, &focal[i].y, &focal[i].start, &focal[i].heat);
			if ( ok != 4 ) {
				fprintf(stderr,"-- Error in file: reading focal point %d from file: %s\n", i, argv[1]);
				exit( EXIT_FAILURE );
			}
			focal[i].active = 0;
		}
	}
	/* 1.3. Read configuration from arguments */
	else {
		/* 1.3.1. Check minimum number of arguments */
		if (argc<6) {
			fprintf(stderr, "-- Error in arguments: not enough arguments when reading configuration from the command line\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}

		/* 1.3.2. Surface and maximum number of iterations */
		rows = atoi( argv[1] );
		columns = atoi( argv[2] );
		max_iter = atoi( argv[3] );

		surface = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );
		surfaceCopy = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );

		/* 1.3.3. Teams information */
		num_teams = atoi( argv[4] );
		teams = (Team *)malloc( sizeof(Team) * (size_t)num_teams );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		if ( argc < num_teams*3 + 5 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_teams; i++ ) {
			teams[i].x = atoi( argv[5+i*3] );
			teams[i].y = atoi( argv[6+i*3] );
			teams[i].type = atoi( argv[7+i*3] );
		}

		/* 1.3.4. Focal points information */
		int focal_args = 5 + i*3;
		if ( argc < focal_args+1 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for the number of focal points\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
		num_focal = atoi( argv[focal_args] );
		focal = (FocalPoint *)malloc( sizeof(FocalPoint) * (size_t)num_focal );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		if ( argc < focal_args + 1 + num_focal*4 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_focal; i++ ) {
			focal[i].x = atoi( argv[focal_args+i*4+1] );
			focal[i].y = atoi( argv[focal_args+i*4+2] );
			focal[i].start = atoi( argv[focal_args+i*4+3] );
			focal[i].heat = atoi( argv[focal_args+i*4+4] );
			focal[i].active = 0;
		}

		/* 1.3.5. Sanity check: No extra arguments at the end of line */
		if ( argc > focal_args+i*4+1 ) {
			fprintf(stderr,"-- Error in arguments: extra arguments at the end of the command line\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
	}


#ifdef DEBUG
	/* 1.4. Print arguments */
	printf("Arguments, Rows: %d, Columns: %d, max_iter: %d\n", rows, columns, max_iter);
	printf("Arguments, Teams: %d, Focal points: %d\n", num_teams, num_focal );
	for( i=0; i<num_teams; i++ ) {
		printf("\tTeam %d, position (%d,%d), type: %d\n", i, teams[i].x, teams[i].y, teams[i].type );
	}
	for( i=0; i<num_focal; i++ ) {
		printf("\tFocal_point %d, position (%d,%d), start time: %d, temperature: %d\n", i,
		focal[i].x,
		focal[i].y,
		focal[i].start,
		focal[i].heat );
	}
#endif // DEBUG

	/* 2. Select GPU and start global timer */
	hipSetDevice(0);
	hipDeviceSynchronize();
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */
 //Declaración de matrices y reserva de memoria del DEVICE
 float *dsurface;
 float *dsurfacecopy;
 float *diferencia;
 float *matAux;
 FocalPoint *focalaux;
 int ITEMS = rows*columns;
 hipError_t errores;
 //float *matAux = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );

 errores=hipMalloc(&focalaux, sizeof(FocalPoint) * num_focal );
	if(errores != hipSuccess){
		printf("Error de Cuda(hipMalloc): %s \n", hipGetErrorString(errores));
	}

 errores = hipMalloc(&matAux, ITEMS * sizeof(float));
 if(errores != hipSuccess){
  printf("Error de Cuda(hipMalloc): %s \n", hipGetErrorString(errores));
 }

 errores = hipMalloc(&dsurface, ITEMS * sizeof(float));
 if(errores != hipSuccess){
 	printf("Error de Cuda(hipMalloc): %s \n", hipGetErrorString(errores));
 }

 errores = hipMalloc(&dsurfacecopy, ITEMS * sizeof(float));
 if(errores != hipSuccess){
 	printf("Error de Cuda(hipMalloc): %s \n", hipGetErrorString(errores));
 }
 errores = hipMalloc((void**)&diferencia, ITEMS * sizeof(float));
 if(errores != hipSuccess){
 	printf("Error de Cuda(hipMalloc): %s \n", hipGetErrorString(errores));
 }

	int hilosx = 32;
	int hilosy = 32;
	int nhilosbloque = hilosx * hilosy;
 	dim3 bloque1(hilosx, hilosy);
 	dim3 grid1( ( (ITEMS) / nhilosbloque ) + 1 );

	/* 3. Initialize surface */
	// for(int gpu = 0, gpu < 4, gpu++){
	//
	// }
 inicializar<<<grid1, bloque1>>>(dsurface, ITEMS);

 // errores = hipDeviceSynchronize();
 // if(errores != hipSuccess){
	//  printf("Error de cuda (hipDeviceSynchronize): %s", hipGetErrorString(errores));
 // }

 errores = hipMemcpy(surface, dsurface, ITEMS * sizeof(float), hipMemcpyDeviceToHost);
 if(errores != hipSuccess){
 	printf("Error de cuda (hipMemcpy): %s", hipGetErrorString(errores));
 }

	// for( i=0; i<rows; i++ )
	// 	for( j=0; j<columns; j++ )
	// 		accessMat( surface, i, j ) = 0.0;

	/* 4. Simulation */
	int iter;
	int flag_stability = 0;
	int first_activation = 0;
	for( iter=0; iter<max_iter && !flag_stability; iter++ ) {

		/* 4.1. Activate focal points */
		int num_deactivated = 0;
		for( i=0; i<num_focal; i++ ) {
			if ( focal[i].start == iter ) {
				focal[i].active = 1;
				if ( ! first_activation ) first_activation = 1;
			}
			// Count focal points already deactivated by a team
			if ( focal[i].active == 2 ) num_deactivated++;
		}

		if(!first_activation) continue; //Saltar primeras iteracciones sin incendios activos

		/* 4.2. Propagate heat (10 steps per each team movement) */
		float global_residual = 0.0f;
		int step;

		errores = hipMemcpy(focalaux,focal,sizeof(FocalPoint)*num_focal,hipMemcpyHostToDevice);
		if(errores != hipSuccess){
			printf("Error de cuda (hipMemcpy): %s", hipGetErrorString(errores));
		}
		errores = hipMemcpy(dsurface, surface, rows*columns*sizeof(float), hipMemcpyHostToDevice);
		if(errores != hipSuccess){
			printf("Error de cuda (hipMemcpy): %s", hipGetErrorString(errores));
		}


		for( step=0; step<10; step++ )	{
			/* 4.2.1. Update heat on active focal points */

			// for( i=0; i<num_focal; i++ ) {
			// 	if ( focal[i].active != 1 ) continue;
			// 	int x = focal[i].x;
			// 	int y = focal[i].y;
			// 	accessMat( surface, x, y ) = focal[i].heat;
			// }
			actualizaCalor<<<grid1,bloque1>>>(dsurface,num_focal,focalaux, columns);

			// errores = hipMemcpy(dsurface, surface, rows*columns*sizeof(float), hipMemcpyHostToDevice);
			// if(errores != hipSuccess){
			// 	printf("Error de cuda (hipMemcpy): %s", hipGetErrorString(errores));
			// }
			// errores = hipMemcpy(dsurfacecopy, surfaceCopy, rows*columns*sizeof(float), hipMemcpyHostToDevice);
			// if(errores != hipSuccess){
			// 	printf("Error de cuda (hipMemcpy): %s", hipGetErrorString(errores));
			// }

			/* 4.2.2. Copy values of the surface in ancillary structure (Skip borders) */
			matAux = dsurface;
			dsurface = dsurfacecopy;
			dsurfacecopy = matAux;

			// for( i=1; i<rows-1; i++ )
			// 	for( j=1; j<columns-1; j++ )
			// 		accessMat( surfaceCopy, i, j ) = accessMat( surface, i, j );

			/* 4.2.3. Update surface values (skip borders) */


			actualizar<<<grid1, bloque1>>>(dsurface, dsurfacecopy, rows, columns);

			//Calculamos matriz diferencia dsurface-dsurfacecpy

			// dim3 bloque1(128);
			// dim3 grid1( ( (ITEMS) / 128 ) + 1 );
			// calc_diferencia<<<grid1, bloque1>>>(dsurface, dsurfacecopy, diferencia, ITEMS);
			// hipDeviceSynchronize();
			// for (int redSize = ITEMS; redSize>1; redSize /= 2) {
			// 	// Agrupa los hilos en bloques de tamaño NUM_THREADS_PER_BLOCK. ¡Cuidado con el último nivel!
			// 	int numBlocks = (redSize/128)+1;
			// 	// Reducción por niveles en la GPU
			// 	calc_global_residual<<< numBlocks, bloque1>>>(diferencia, ITEMS );
			//
			// }


			// hipMemcpy(&global_residual, diferencia, sizeof(float), hipMemcpyDeviceToHost);
			// if(errores != hipSuccess){
			// 	printf("Error de cuda (hipMemcpy): %s", hipGetErrorString(errores));
			// }




			// for( i=1; i<rows-1; i++ )
			// 	for( j=1; j<columns-1; j++ )
			// 		accessMat( surface, i, j ) = (
			// 			accessMat( surfaceCopy, i-1, j ) +
			// 			accessMat( surfaceCopy, i+1, j ) +
			// 			accessMat( surfaceCopy, i, j-1 ) +
			// 			accessMat( surfaceCopy, i, j+1 ) ) / 4;

			/* 4.2.4. Compute the maximum residual difference (absolute value) */
			if (step == 0 || step == 9 ){
				//Calculamos el global_residual desde un kernel
				calc_diferencia<<<grid1, bloque1>>>(dsurface, dsurfacecopy, diferencia, ITEMS);
				errores = hipDeviceSynchronize();
			  if(errores != hipSuccess){
			 	 printf("Error de cuda (hipDeviceSynchronize): %s", hipGetErrorString(errores));
			  }

				for (int redSize = ITEMS; redSize>1; redSize /= 2) {
					// Agrupa los hilos en bloques de tamaño NUM_THREADS_PER_BLOCK. ¡Cuidado con el último nivel!
					int numBlocks = (redSize/nhilosbloque)+1;
					// Reducción por niveles en la GPU
					calc_global_residual<<< numBlocks, bloque1>>>(diferencia, redSize, columns );

				}
				//calc_global_residual<<< 1, 1>>>(diferencia, 3, columns );
				// errores = hipDeviceSynchronize();
			  // if(errores != hipSuccess){
			 	//  printf("Error de cuda (hipDeviceSynchronize): %s", hipGetErrorString(errores));
			  // }
				float glob_resi;
				hipMemcpy(&glob_resi, diferencia, sizeof(float), hipMemcpyDeviceToHost);
				if(errores != hipSuccess){
					printf("Error de cuda (hipMemcpy): %s", hipGetErrorString(errores));
				}
				if (glob_resi > global_residual){
					global_residual = glob_resi;
				}
				

				//Calculamos el global_residual desde la CPU
			// 	errores = hipMemcpy(surface, dsurface, rows*columns*sizeof(float), hipMemcpyDeviceToHost);
			// 	if(errores != hipSuccess){
			// 		printf("Error de cuda (hipMemcpy): %s", hipGetErrorString(errores));
			// 	}
			//
			// 	errores = hipMemcpy(surfaceCopy, dsurfacecopy, rows*columns*sizeof(float), hipMemcpyDeviceToHost);
			// 	if(errores != hipSuccess){
			// 		printf("Error de cuda (hipMemcpy): %s", hipGetErrorString(errores));
			// 	}
			// 	for( i=1; i<rows-1; i++ )
			// 		for( j=1; j<columns-1; j++ )
			// 			if ( fabs( accessMat( surface, i, j ) - accessMat( surfaceCopy, i, j ) ) > global_residual ) {
			// 				global_residual = fabs( accessMat( surface, i, j ) - accessMat( surfaceCopy, i, j ) );
			// 			}
			// 	if (glob_resi == global_residual){
			// 		printf("Test PASSED [iter: %d - step: %d] (%f) == (%f) %d\n", iter, step, glob_resi, global_residual, nhilosbloque );
			// 	}else{
			// 		printf("Test FAILED [iter: %d - step: %d] (%f) != (%f) %d\n", iter, step, glob_resi, global_residual, nhilosbloque );
			// 		exit(EXIT_FAILURE);
			// }
				}
		}

			errores = hipMemcpy(surface, dsurface, rows*columns*sizeof(float), hipMemcpyDeviceToHost);
			if(errores != hipSuccess){
				printf("Error de cuda (hipMemcpy): %s", hipGetErrorString(errores));
			}

			errores = hipMemcpy(surfaceCopy, dsurfacecopy, rows*columns*sizeof(float), hipMemcpyDeviceToHost);
			if(errores != hipSuccess){
				printf("Error de cuda (hipMemcpy): %s", hipGetErrorString(errores));
			}

		/* If the global residual is lower than THRESHOLD, we have reached enough stability, stop simulation at the end of this iteration */
		if( num_deactivated == num_focal && global_residual < THRESHOLD ){
			flag_stability = 1;
		}

		/* 4.3. Move teams */

		for( t=0; t<num_teams; t++ ) {
			/* 4.3.1. Choose nearest focal point */
			float distance = FLT_MAX;
			int target = -1;
			for( j=0; j<num_focal; j++ ) {
				if ( focal[j].active != 1 ) continue; // Skip non-active focal points
				float dx = focal[j].x - teams[t].x;
				float dy = focal[j].y - teams[t].y;
				float local_distance = sqrtf( dx*dx + dy*dy );
				if ( local_distance < distance ) {
					distance = local_distance;
					target = j;
				}
			}
			/* 4.3.2. Annotate target for the next stage */
			teams[t].target = target;

			/* 4.3.3. No active focal point to choose, no movement */
			if ( target == -1 ) continue;

			/* 4.3.4. Move in the focal point direction */
			switch(teams[t].type){
				case 1:
					// Type 1: Can move in diagonal
					if ( focal[target].x < teams[t].x ) teams[t].x--;
					if ( focal[target].x > teams[t].x ) teams[t].x++;
					if ( focal[target].y < teams[t].y ) teams[t].y--;
					if ( focal[target].y > teams[t].y ) teams[t].y++;
					break;
				case 2:
					// Type 2: First in horizontal direction, then in vertical direction
					if ( focal[target].y < teams[t].y ) teams[t].y--;
					else if ( focal[target].y > teams[t].y ) teams[t].y++;
					else if ( focal[target].x < teams[t].x ) teams[t].x--;
					else if ( focal[target].x > teams[t].x ) teams[t].x++;
					break;
				case 3:
					// Type 3: First in vertical direction, then in horizontal direction
					if ( focal[target].x < teams[t].x ) teams[t].x--;
					else if ( focal[target].x > teams[t].x ) teams[t].x++;
					else if ( focal[target].y < teams[t].y ) teams[t].y--;
					else if ( focal[target].y > teams[t].y ) teams[t].y++;
					break;
			}
		}

		/* 4.4. Team actions */
		for( t=0; t<num_teams; t++ ) {
			/* 4.4.1. Deactivate the target focal point when it is reached */
			int target = teams[t].target;
			if ( target != -1 && focal[target].x == teams[t].x && focal[target].y == teams[t].y
				&& focal[target].active == 1 )
				focal[target].active = 2;

			/* 4.4.2. Reduce heat in a circle around the team */
			int radius;
			// Influence area of fixed radius depending on type
			if ( teams[t].type == 1 ) radius = RADIUS_TYPE_1;
			else radius = RADIUS_TYPE_2_3;
			for( i=teams[t].x-radius; i<=teams[t].x+radius; i++ ) {
				for( j=teams[t].y-radius; j<=teams[t].y+radius; j++ ) {
					if ( i<1 || i>=rows-1 || j<1 || j>=columns-1 ) continue; // Out of the heated surface
					float dx = teams[t].x - i;
					float dy = teams[t].y - j;
					float distance = sqrtf( dx*dx + dy*dy );
					if ( distance <= radius ) {
						accessMat( surface, i, j ) = accessMat( surface, i, j ) * ( 0.75 ); // Team efficiency factor
					}
				}
			}
		}

#ifdef DEBUG
		/* 4.5. DEBUG: Print the current state of the simulation at the end of each iteration */
		print_status( iter, rows, columns, surface, num_teams, teams, num_focal, focal, global_residual );
#endif // DEBUG
	}

	//Liberamos memoria del DEVICE
	errores = hipFree(dsurface);
	if(errores != hipSuccess){
		printf("Error de cuda (hipFree): %s", hipGetErrorString(errores));
	}
	errores = hipFree(dsurfacecopy);
	if(errores != hipSuccess){
		printf("Error de cuda (hipFree): %s", hipGetErrorString(errores));
	}

/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	/* 5. Stop global time */
	hipDeviceSynchronize();
	ttotal = cp_Wtime() - ttotal;

	/* 6. Output for leaderboard */
	printf("\n");
	/* 6.1. Total computation time */
	printf("Time: %lf\n", ttotal );
	/* 6.2. Results: Number of iterations, position of teams, residual heat on the focal points */
	printf("Result: %d", iter);
	/*
	for (i=0; i<num_teams; i++)
		printf(" %d %d", teams[i].x, teams[i].y );
	*/
	for (i=0; i<num_focal; i++)
		printf(" %.6f", accessMat( surface, focal[i].x, focal[i].y ) );
	printf("\n");

	/* 7. Free resources */
	free( teams );
	free( focal );
	free( surface );
	free( surfaceCopy );

	/* 8. End */
	return 0;
}
