#include "hip/hip_runtime.h"
/*
 * Simplified simulation of life evolution
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2019/2020
 *
 * v1.5
 *
 * CHANGES:
 * 1) Float values have been substituted by fixed point arithmetics 
 *	using integers. To simplify, the fixed point arithmetics are done 
 *	with PRECISION in base 10. See precision constant in int_float.h
 * 2) It uses a portable approximation to trigonometric functions using
 *	Taylor polynomials. 
 * 3) nrand48 function has been extracted from glibc source code and 
 *	its internal API simplified to allow its use in the GPU.
 *
 * (c) 2020, Arturo Gonzalez Escribano
 */
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<stdbool.h>
#include<cputils.h>
#include<hip/hip_runtime.h>
#include<int_float.h>

/* 
 * Constants: Converted to fixed point with the given PRECISION
 */
#define ENERGY_NEEDED_TO_LIVE		PRECISION / 10	// Equivalent to 0.1
#define ENERGY_NEEDED_TO_MOVE		PRECISION	// Equivalent to 1.0
#define ENERGY_SPENT_TO_LIVE		PRECISION / 5	// Equivalent to 0.2
#define ENERGY_SPENT_TO_MOVE		PRECISION	// Equivalent to 1.0
#define ENERGY_NEEDED_TO_SPLIT		PRECISION * 20	// Equivalent to 20.0


/* Structure to store data of a cell */
typedef struct {
	int pos_row, pos_col;		// Position
	int mov_row, mov_col;		// Direction of movement
	int choose_mov[3];		// Genes: Probabilities of 0 turning-left; 1 advance; 2 turning-right
	int storage;			// Food/Energy stored
	int age;			// Number of steps that the cell has been alive
	unsigned short random_seq[3];	// Status value of its particular random sequence
	bool alive;			// Flag indicating if the cell is still alive
} Cell;


/* Structure for simulation statistics */
typedef struct {
	int history_total_cells;	// Accumulated number of cells created
	int history_dead_cells;		// Accumulated number of dead cells
	int history_max_alive_cells;	// Maximum number of cells alive in a step
	int history_max_new_cells;	// Maximum number of cells created in a step
	int history_max_dead_cells;	// Maximum number of cells died in a step
	int history_max_age;		// Maximum age achieved by a cell
	int history_max_food;		// Maximum food level in a position of the culture
} Statistics;


/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 *	USE THIS SPACE FOR YOUR KERNEL OR DEVICE FUNTIONS
 *
 */

#include "taylor_trig.h"
#include "glibc_nrand48.h"
#include "evolution_kernels.h"

/*
 * Struct for a pair position-food, for food generation
 */
#ifndef FOOD_T
#define FOOD_T
typedef struct {
	int pos;
	int food;
} food_t;
#endif

/*
 * Get an uniformly distributed random number between 0 and max
 * It uses glibc_nrand, that returns a number between 0 and 2^31
 */
#define int_urand48( max, seq )	(int)( (long)(max) * glibc_nrand48( seq ) / 2147483648 )

/* 
 * Macro function to simplify accessing with two coordinates to a flattened array
 * 	This macro-function can be modified by the students if needed
 *
 */
#define accessMat( arr, exp1, exp2 )	arr[ (int)(exp1) * columns + (int)(exp2) ]
#define matPos(exp1, exp2)	(int)(exp1) / PRECISION * columns + (int)(exp2) / PRECISION

/*
 * Function: Choose a new direction of movement for a cell
 * 	This function can be changed and/or optimized by the students
 */
__host__ __device__ void cell_new_direction( Cell *cell ) {
	int angle = int_urand48( INT_2PI, cell->random_seq );
	cell->mov_row = taylor_sin( angle );
	cell->mov_col = taylor_cos( angle );
}

/*
 * Function: Mutation of the movement genes on a new cell
 * 	This function can be changed and/or optimized by the students
 */
__host__ __device__ void cell_mutation( Cell *cell ) {
	/* 1. Select which genes change:
	 	0 Left grows taking part of the Advance part
	 	1 Advance grows taking part of the Left part
	 	2 Advance grows taking part of the Right part
	 	3 Right grows taking part of the Advance part
	*/
	int mutation_type = int_urand48( 4, cell->random_seq );
	/* 2. Select the amount of mutation (up to 50%) */
	int mutation_percentage = int_urand48( PRECISION / 2, cell->random_seq );
	/* 3. Apply the mutation */
	int mutation_value;
	switch( mutation_type ) {
		case 0:
			mutation_value = intfloatMult( cell->choose_mov[1] , mutation_percentage );
			cell->choose_mov[1] -= mutation_value;
			cell->choose_mov[0] += mutation_value;
			break;
		case 1:
			mutation_value = intfloatMult( cell->choose_mov[0] , mutation_percentage );
			cell->choose_mov[0] -= mutation_value;
			cell->choose_mov[1] += mutation_value;
			break;
		case 2:
			mutation_value = intfloatMult( cell->choose_mov[2] , mutation_percentage );
			cell->choose_mov[2] -= mutation_value;
			cell->choose_mov[1] += mutation_value;
			break;
		case 3:
			mutation_value = intfloatMult( cell->choose_mov[1] , mutation_percentage );
			cell->choose_mov[1] -= mutation_value;
			cell->choose_mov[2] += mutation_value;
			break;
	}
	/* 4. Correct potential precision problems */
	cell->choose_mov[2] = PRECISION - cell->choose_mov[1] - cell->choose_mov[0];
}

/*
 * CUDA block reduction
 * Inputs: 
 *	Device pointer to an array of int of any size
 *	Size of the array
 *	Device pointer to an int to store the result
 * 
 * Launching parameters:
 *	One-dimesional grid of any size
 *	Any valid block size
 *	Dynamic shared memory size equal to: sizeof(int) * block size
 *
 * (c) 2020, Arturo Gonzalez-Escribano
 * Simplification for an assignment in a Parallel Computing course,
 * Computing Engineering Degree, Universidad de Valladolid
 * Academic year 2019/2020
 */
__global__ void reductionMax(int* array, int size, int *result)
{
	int tid = threadIdx.x;
	int globalPos = tid + blockIdx.x * blockDim.x;

	extern __shared__ int buffer[ ];
	if ( globalPos < size ) { 
		buffer[ tid ] = array[ globalPos ];
	}
	else buffer[ tid ] = 0;
	__syncthreads();

	// Each iteration half of the buffer elements are reduced.
	for( int step=blockDim.x/2; step>=1; step /= 2 ) {
		if ( tid < step )
			if ( buffer[ tid ] < buffer[ tid + step ] )
				buffer[ tid ] = buffer[ tid + step ];
		if ( step > 32 )	// Warps don't need to be synced explicitly.
			__syncthreads();
	}
	// TODO: unroll warp?

	if ( tid == 0 )
		atomicMax( result, buffer[0] );
}


/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */


#ifdef DEBUG
/* 
 * Function: Print the current state of the simulation 
 */
void print_status( int iteration, int rows, int columns, int *culture, int num_cells, Cell *cells, int num_cells_alive, Statistics sim_stat ) {
	/* 
	 * You don't need to optimize this function, it is only for pretty printing and debugging purposes.
	 * It is not compiled in the production versions of the program.
	 * Thus, it is never used when measuring times in the leaderboard
	 */
	int i,j;

	printf("Iteration: %d\n", iteration );
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	for( i=0; i<rows; i++ ) {
		printf("|");
		for( j=0; j<columns; j++ ) {
			char symbol;
			if ( accessMat( culture, i, j ) >= 20 * PRECISION ) symbol = '+';
			else if ( accessMat( culture, i, j ) >= 10 * PRECISION ) symbol = '*';
			else if ( accessMat( culture, i, j ) >= 5 * PRECISION ) symbol = '.';
			else symbol = ' ';

			int t;
			int counter = 0;
			for( t=0; t<num_cells; t++ ) {
				int row = (int)(cells[t].pos_row / PRECISION);
				int col = (int)(cells[t].pos_col / PRECISION);
				if ( cells[t].alive && row == i && col == j ) {
					counter ++;
				}
			}
			if ( counter > 9 ) printf("(M)" );
			else if ( counter > 0 ) printf("(%1d)", counter );
			else printf(" %c ", symbol );
		}
		printf("|\n");
	}
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	printf("Num_cells_alive: %04d\nHistory( Cells: %04d, Dead: %04d, Max.alive: %04d, Max.new: %04d, Max.dead: %04d, Max.age: %04d, Max.food: %6f )\n\n", 
		num_cells_alive, 
		sim_stat.history_total_cells, 
		sim_stat.history_dead_cells, 
		sim_stat.history_max_alive_cells, 
		sim_stat.history_max_new_cells, 
		sim_stat.history_max_dead_cells, 
		sim_stat.history_max_age,
		(float)sim_stat.history_max_food / PRECISION
	);
}
#endif

/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s ", program_name );
	fprintf(stderr,"<rows> <columns> <maxIter> <max_food> <food_density> <food_level> <short_rnd1> <short_rnd2> <short_rnd3> <num_cells>\n");
	fprintf(stderr,"\tOptional arguments for special food spot: [ <row> <col> <size_rows> <size_cols> <density> <level> ]\n");
	fprintf(stderr,"\n");
}


/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	int i,j;

	// Simulation data
	int max_iter;			// Maximum number of simulation steps
	int rows, columns;		// Cultivation area sizes
	int *culture;			// Cultivation area values
	int *culture_cells;		// Ancillary structure to count the number of cells in a culture space

	float max_food;			// Maximum level of food on any position
	float food_density;		// Number of food sources introduced per step
	float food_level;		// Maximum number of food level in a new source

	bool food_spot_active = false;	// Special food spot: Active
	int food_spot_row = 0;		// Special food spot: Initial row
	int food_spot_col = 0;		// Special food spot: Initial row
	int food_spot_size_rows = 0;	// Special food spot: Rows size
	int food_spot_size_cols = 0;	// Special food spot: Cols size
	float food_spot_density = 0.0f;	// Special food spot: Food density
	float food_spot_level = 0.0f;	// Special food spot: Food level

	unsigned short init_random_seq[3];	// Status of the init random sequence
	unsigned short food_random_seq[3];	// Status of the food random sequence
	unsigned short food_spot_random_seq[3];	// Status of the special food spot random sequence

	int	num_cells;		// Number of cells currently stored in the list
	Cell	*cells;			// List to store cells information

	// Statistics
	Statistics sim_stat;	
	sim_stat.history_total_cells = 0;
	sim_stat.history_dead_cells = 0;
	sim_stat.history_max_alive_cells = 0;
	sim_stat.history_max_new_cells = 0;
	sim_stat.history_max_dead_cells = 0;
	sim_stat.history_max_age = 0;
	sim_stat.history_max_food = 0.0f;

	/* 1. Read simulation arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc < 11) {
		fprintf(stderr, "-- Error: Not enough arguments when reading configuration from the command line\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	/* 1.2. Read culture sizes, maximum number of iterations */
	rows = atoi( argv[1] );
	columns = atoi( argv[2] );
	max_iter = atoi( argv[3] );

	/* 1.3. Food data */
	max_food = atof( argv[4] );
	food_density = atof( argv[5] );
	food_level = atof( argv[6] );

	/* 1.4. Read random sequences initializer */
	for( i=0; i<3; i++ ) {
		init_random_seq[i] = (unsigned short)atoi( argv[7+i] );
	}

	/* 1.5. Read number of cells */
	num_cells = atoi( argv[10] );

	/* 1.6. Read special food spot */
	if (argc > 11 ) {
		if ( argc < 17 ) {
			fprintf(stderr, "-- Error in number of special-food-spot arguments in the command line\n\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
		else {
			food_spot_active = true;
			food_spot_row = atoi( argv[11] );
			food_spot_col = atoi( argv[12] );
			food_spot_size_rows = atoi( argv[13] );
			food_spot_size_cols = atoi( argv[14] );
			food_spot_density = atof( argv[15] );
			food_spot_level = atof( argv[16] );

			// Check non-used trailing arguments
			if ( argc > 17 ) {
				fprintf(stderr, "-- Error: too many arguments in the command line\n\n");
				show_usage( argv[0] );
				exit( EXIT_FAILURE );
			}
		}
	}

#ifdef DEBUG
	/* 1.7. Print arguments */
	printf("Arguments, Rows: %d, Columns: %d, max_iter: %d\n", rows, columns, max_iter);
	printf("Arguments, Max.food: %f, Food density: %f, Food level: %f\n", max_food, food_density, food_level);
	printf("Arguments, Init Random Sequence: %hu,%hu,%hu\n", init_random_seq[0], init_random_seq[1], init_random_seq[2]);
	if ( food_spot_active ) {
		printf("Arguments, Food_spot, pos(%d,%d), size(%d,%d), Density: %f, Level: %f\n",
			food_spot_row, food_spot_col, food_spot_size_rows, food_spot_size_cols, food_spot_density, food_spot_level );
	}
	printf("Initial cells: %d\n", num_cells );
#endif // DEBUG


	/* 1.8. Initialize random sequences for food dropping */
	for( i=0; i<3; i++ ) {
		food_random_seq[i] = (unsigned short)glibc_nrand48( init_random_seq );
		food_spot_random_seq[i] = (unsigned short)glibc_nrand48( init_random_seq );
	}

	/* 1.9. Initialize random sequences of cells */
	cells = (Cell *)malloc( sizeof(Cell) * (size_t)num_cells );
	if ( cells == NULL ) {
		fprintf(stderr,"-- Error allocating: %d cells\n", num_cells );
		exit( EXIT_FAILURE );
	}
	for( i=0; i<num_cells; i++ ) {
		// Initialize the cell ramdom sequences
		for( j=0; j<3; j++ ) 
			cells[i].random_seq[j] = (unsigned short)glibc_nrand48( init_random_seq );
	}


#ifdef DEBUG
	/* 1.10. Print random seed of the initial cells */
	/*
	printf("Initial cells random seeds: %d\n", num_cells );
	for( i=0; i<num_cells; i++ )
		printf("\tCell %d, Random seq: %hu,%hu,%hu\n", i, cells[i].random_seq[0], cells[i].random_seq[1], cells[i].random_seq[2] );
	*/
#endif // DEBUG


	// CUDA start
	hipSetDevice(0);
	hipDeviceSynchronize();

	/* 2. Start global timer */
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */

#include "cuda_check.h"
#include "cuda_time.h"

	/* 3. Initialize culture surface and initial cells */
	cudaCheckCall((hipMalloc(&culture, sizeof(int) * (size_t)rows * (size_t)columns)));
	cudaCheckCall((hipMalloc(&culture_cells, sizeof(int) * (size_t)rows * (size_t)columns)));

	// 3.1
	time_start();
	cudaCheckCall((hipMemset(culture, 0, sizeof(int) * (size_t)rows * (size_t)columns)));
	cudaCheckCall((hipMemset(culture_cells, 0, sizeof(int) * (size_t)rows * (size_t)columns)));
	time_end(time3_1);

	// 3.2
	time_start();
	for( i=0; i<num_cells; i++ ) {
		cells[i].alive = true;
		// Initial age: Between 1 and 20 
		cells[i].age = 1 + int_urand48( 19, cells[i].random_seq );
		// Initial storage: Between 10 and 20 units
		cells[i].storage = 10 * PRECISION + int_urand48( 10 * PRECISION, cells[i].random_seq );
		// Initial position: Anywhere in the culture arena
		cells[i].pos_row = int_urand48( rows * PRECISION, cells[i].random_seq );
		cells[i].pos_col = int_urand48( columns * PRECISION, cells[i].random_seq );
		// Movement direction: Unity vector in a random direction
		cell_new_direction( &cells[i] );
		// Movement genes: Probabilities of advancing or changing direction: The sum should be 1.00
		cells[i].choose_mov[0] = PRECISION / 3;
		cells[i].choose_mov[2] = PRECISION / 3;
		cells[i].choose_mov[1] = PRECISION - cells[i].choose_mov[0] - cells[i].choose_mov[2];
	}
	Cell *cells_device;
	cudaCheckCall((hipMalloc(&cells_device, sizeof(Cell) * (size_t)num_cells)));
	cudaCheckCall((hipMemcpy(cells_device, cells, num_cells, hipMemcpyHostToDevice)));

	// Statistics: Initialize total number of cells, and max. alive
	sim_stat.history_total_cells = num_cells;
	sim_stat.history_max_alive_cells = num_cells;
	time_end(time3_2);

#ifdef DEBUG
	/* Show initial cells data */
	printf("Initial cells data: %d\n", num_cells );
	for( i=0; i<num_cells; i++ ) {
		printf("\tCell %d, Pos(%f,%f), Mov(%f,%f), Choose_mov(%f,%f,%f), Storage: %f, Age: %d\n",
				i, 
				(float)cells[i].pos_row / PRECISION, 
				(float)cells[i].pos_col / PRECISION, 
				(float)cells[i].mov_row / PRECISION, 
				(float)cells[i].mov_col / PRECISION, 
				(float)cells[i].choose_mov[0] / PRECISION, 
				(float)cells[i].choose_mov[1] / PRECISION, 
				(float)cells[i].choose_mov[2] / PRECISION, 
				(float)cells[i].storage / PRECISION,
				cells[i].age );
	}
#endif // DEBUG

	/* 4. Simulation */
	int current_max_food = 0;
	int num_cells_alive = num_cells;
	int iter;
	int max_food_int = max_food * PRECISION;

	int num_new_sources = (int)(rows * columns * food_density);
	int num_new_sources_spot = food_spot_active ? (int)(food_spot_size_rows * food_spot_size_cols * food_spot_density) : 0;
	int max_new_sources = max(num_new_sources, num_new_sources_spot);

	food_t *food_spots = (food_t *)malloc(sizeof(food_t) * max_new_sources);
	food_t *food_spots_device;
	cudaCheckCall((hipMalloc(&food_spots_device, sizeof(food_t) * max_new_sources)));

	for( iter=0; iter<max_iter && current_max_food <= max_food_int && num_cells_alive > 0; iter++ ) {
		printf("Iteracion %d\n",iter);
		update_times();

		int step_new_cells = 0;
		int step_dead_cells = 0;

		/* 4.1. Spreading new food */
		time_start();
		// Across the whole culture
		for (i=0; i<num_new_sources; i++) {
			int row = int_urand48( rows, food_random_seq );
			food_spots[i].pos = row*columns;
			int col = int_urand48( columns, food_random_seq );
			food_spots[i].pos += col;
			food_spots[i].food = int_urand48( food_level * PRECISION, food_random_seq );
		}
		printf("num_new_sources: %d\n",num_new_sources);
		cudaCheckCall((hipMemcpy(food_spots_device, food_spots, sizeof(food_t), hipMemcpyHostToDevice))); //AQUI!
		printf("fium\n");
		cudaCheckKernel((placeFood<<<num_new_sources/1024 + 1, 1024>>>(culture, food_spots_device, num_new_sources)));
		// In the special food spot
		if ( food_spot_active ) {
			for (i=0; i<num_new_sources_spot; i++) {
				int row = (food_spot_row + int_urand48( food_spot_size_rows, food_spot_random_seq ));
				food_spots[i].pos = row*columns;
				int col = food_spot_col + int_urand48( food_spot_size_cols, food_spot_random_seq );
				food_spots[i].pos += col;
				food_spots[i].food = int_urand48( food_spot_level * PRECISION, food_spot_random_seq );
			}
			cudaCheckCall((hipMemcpy(food_spots_device, food_spots, sizeof(food_t) * num_new_sources_spot, hipMemcpyHostToDevice)));
			cudaCheckKernel((placeFood<<<num_new_sources_spot/1024 + 1, 1024>>>(culture, food_spots_device, num_new_sources_spot)));
		}
		time_end(time4_1);

		/* 4.2. Prepare ancillary data structures */
		time_start();
		/* 4.2.1. Clear ancillary structure of the culture to account alive cells in a position after movement */		
		cudaCheckCall((hipMemset(culture_cells, 0, sizeof(int) * (size_t)rows * (size_t)columns)));
		time_end(time4_2);

		/* 4.3. Cell movements */
		time_start();
		for (i=0; i<num_cells; i++) {
			if ( cells[i].alive ) {
				cells[i].age ++;
				// Statistics: Max age of a cell in the simulation history
				if ( cells[i].age > sim_stat.history_max_age ) sim_stat.history_max_age = cells[i].age;

				/* 4.3.1. Check if the cell has the needed energy to move or keep alive */
				if ( cells[i].storage < ENERGY_NEEDED_TO_LIVE ) {
					// Cell has died
					cells[i].alive = false;
					num_cells_alive --;
					step_dead_cells ++;
					continue;
				}
				if ( cells[i].storage < ENERGY_NEEDED_TO_MOVE ) {
					// Almost dying cell, it cannot move, only if enough food is dropped here it will survive
					cells[i].storage -= ENERGY_SPENT_TO_LIVE;
				}
				else {
					// Consume energy to move
					cells[i].storage -= ENERGY_SPENT_TO_MOVE;
						
					/* 4.3.2. Choose movement direction */
					int prob = int_urand48( PRECISION, cells[i].random_seq );
					if ( prob < cells[i].choose_mov[0] ) {
						// Turn left (90 degrees)
						int tmp = cells[i].mov_col;
						cells[i].mov_col = cells[i].mov_row;
						cells[i].mov_row = -tmp;
					}
					else if ( prob >= cells[i].choose_mov[0] + cells[i].choose_mov[1] ) {
						// Turn right (90 degrees)
						int tmp = cells[i].mov_row;
						cells[i].mov_row = cells[i].mov_col;
						cells[i].mov_col = -tmp;
					}
					// else do not change the direction
					
					/* 4.3.3. Update position moving in the choosen direction*/
					cells[i].pos_row += cells[i].mov_row;
					cells[i].pos_col += cells[i].mov_col;
					// Periodic arena: Left/Rigth edges are connected, Top/Bottom edges are connected
					if ( cells[i].pos_row < 0 ) cells[i].pos_row += rows * PRECISION;
					if ( cells[i].pos_row >= rows * PRECISION) cells[i].pos_row -= rows * PRECISION;
					if ( cells[i].pos_col < 0 ) cells[i].pos_col += columns * PRECISION;
					if ( cells[i].pos_col >= columns * PRECISION) cells[i].pos_col -= columns * PRECISION;
				}

				/* 4.3.4. Annotate that there is one more cell in this culture position */
				cudaCheckKernel((addInDeviceArray<<<1, 1>>>(culture_cells, matPos(cells[i].pos_row, cells[i].pos_col), 1)));
			}
		} // End cell movements
		time_end(time4_3);

		/* 4.6. Clean dead cells from the original list */
		time_start();
		// 4.6.1. Move alive cells to the left to substitute dead cells
		int free_position = 0;
		for( i=0; i<num_cells; i++ ) {
			if ( cells[i].alive ) {
				if ( free_position != i ) {
					cells[free_position] = cells[i];
				}
				free_position ++;
			}
		}
		// 4.6.2. Reduce the storage space of the list to the current number of cells
		num_cells = free_position;
		cells = (Cell *)realloc( cells, sizeof(Cell) * num_cells );
		time_end(time4_6);

		// Expand cell list:
		Cell *new_cells;
		cudaCheckCall((hipMalloc(&new_cells, sizeof(Cell) * 2 * num_cells)));
		cudaCheckCall((hipMemset(new_cells, 0, sizeof(Cell) * 2 * num_cells_alive)));
		cudaCheckCall((hipMemcpy(new_cells, cells, sizeof(Cell) * num_cells_alive,hipMemcpyDeviceToHost))); //no iría más abajo?
		cudaCheckCall((hipFree(cells)));
		cells = new_cells;

		int *step_new_cells_device;
		cudaCheckCall((hipMalloc(&step_new_cells_device, sizeof(int))));
		step_new_cells_device = (int *)malloc(sizeof(int));
		cudaCheckKernel((evolution44_45<<<num_cells + 1,1024>>>(culture,culture_cells,columns,num_cells, cells_device,step_new_cells_device)));
		fprintf(stderr,"Hago bien la operacion\n");
		cudaCheckCall((hipMemcpy(&step_new_cells, step_new_cells_device, sizeof(int), hipMemcpyDeviceToHost)));
		cudaCheckCall((hipMemcpy(cells,cells_device,sizeof(Cell)*(num_cells+step_new_cells),hipMemcpyDeviceToHost))); //devolver cells
		fprintf(stderr,"Hago bien em memcpy\n");
		// History y num_cells_alive

		/* 4.8. Decrease non-harvested food */
		time_start();
		cudaCheckKernel((foodDecrease<<<rows*columns/1024 + 1, 1024>>>(culture, rows*columns)));

		int *current_max_food_device;
		cudaCheckCall((hipMalloc(&current_max_food_device, sizeof(int))));	// TODO ???
		cudaCheckKernel((reductionMax<<<rows*columns/1024 + 1, 1024, sizeof(int) * 1024>>>(culture, rows*columns, current_max_food_device)));

		cudaCheckCall((hipMemcpy(&current_max_food, current_max_food_device, sizeof(int), hipMemcpyDeviceToHost)));
		time_end(time4_8);
		fprintf(stderr,"fin\n");


		cudaCheckCall((hipMemcpy(food_spots, food_spots_device, sizeof(food_t), hipMemcpyDeviceToHost))); //devolver food_spots

		/* 4.9. Statistics */
		time_start();
		// Statistics: Max food
		if ( current_max_food > sim_stat.history_max_food ) sim_stat.history_max_food = current_max_food;
		// Statistics: Max new cells per step
		if ( step_new_cells > sim_stat.history_max_new_cells ) sim_stat.history_max_new_cells = step_new_cells;
		// Statistics: Accumulated dead and Max dead cells per step
		sim_stat.history_dead_cells += step_dead_cells;
		if ( step_dead_cells > sim_stat.history_max_dead_cells ) sim_stat.history_max_dead_cells = step_dead_cells;
		// Statistics: Max alive cells per step
		if ( num_cells_alive > sim_stat.history_max_alive_cells ) sim_stat.history_max_alive_cells = num_cells_alive;
		time_end(time4_9);

#ifdef DEBUG
		/* 4.10. DEBUG: Print the current state of the simulation at the end of each iteration */
		print_status( iter, rows, columns, culture, num_cells, cells, num_cells_alive, sim_stat );
#endif // DEBUG
	}

	hipFree(culture);
	hipFree(culture_cells);
	culture = culture_cells = NULL;	// Avoid the segmentation fault later.
	print_times();
	
/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	// CUDA stop
	hipDeviceSynchronize();

	/* 5. Stop global time */
	ttotal = cp_Wtime() - ttotal;

#ifdef DEBUG
	printf("List of cells at the end of the simulation: %d\n\n", num_cells );
	for( i=0; i<num_cells; i++ ) {
		printf("Cell %d, Alive: %d, Pos(%f,%f), Mov(%f,%f), Choose_mov(%f,%f,%f), Storage: %f, Age: %d\n",
				i,
				cells[i].alive,
				(float)cells[i].pos_row / PRECISION, 
				(float)cells[i].pos_col / PRECISION, 
				(float)cells[i].mov_row / PRECISION, 
				(float)cells[i].mov_col / PRECISION, 
				(float)cells[i].choose_mov[0] / PRECISION, 
				(float)cells[i].choose_mov[1] / PRECISION, 
				(float)cells[i].choose_mov[2] / PRECISION, 
				(float)cells[i].storage / PRECISION,
				cells[i].age );
	}
#endif // DEBUG

	/* 6. Output for leaderboard */
	printf("\n");
	/* 6.1. Total computation time */
	printf("Time: %lf\n", ttotal );

	/* 6.2. Results: Number of iterations and other statistics */
	printf("Result: %d, ", iter);
	printf("%d, %d, %d, %d, %d, %d, %d, %f\n", 
		num_cells_alive, 
		sim_stat.history_total_cells, 
		sim_stat.history_dead_cells, 
		sim_stat.history_max_alive_cells, 
		sim_stat.history_max_new_cells, 
		sim_stat.history_max_dead_cells, 
		sim_stat.history_max_age,
		(float)sim_stat.history_max_food / PRECISION
	);

	/* 7. Free resources */	
	free( culture );
	free( culture_cells );
	free( cells );

	/* 8. End */
	return 0;
}
